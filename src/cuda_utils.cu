#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "cuda_utils.cuh"

__host__ void random_init(data_t* data,
                          size_t size_in_bytes,
                          unsigned long long seed) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerate(gen, (unsigned int*)data,
                   size_in_bytes / sizeof(unsigned int));
    hiprandDestroyGenerator(gen);
}
